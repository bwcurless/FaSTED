#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <math.h>
#include <hip/hip_vector_types.h>

#include <iostream>
#include <vector>

#include "blockMma.cuh"
#include "utils.cuh"

using SharedSize = WarpMma::SharedSize;
using InPrec = Mma::InPrec;

__global__ void MmaPtxShared(unsigned long long* iterationCount, SharedSize* AValues,
                             SharedSize* BValues, int kStride);
__device__ uint get_smid(void);

// ---------- Matrix Parameters ----------
constexpr int numPoints = 1024 * 16;
constexpr Mma::mmaShape globalMmaShape{numPoints, numPoints, 64 * 64};

// ---------- Mma parameters ----------

// ---------- Warp parameters ----------

// ---------- Hardware parameters ----------

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Return the ID of the steaming multiprocesser this block is running on
__device__ uint get_smid(void) {
    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret));

    return ret;
}

int main(int argc, char* argv[]) {
    hipSetDevice(0);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate this just so kernel actually does something
    unsigned long long* d_iterationCount;
    unsigned long long h_iterationCount = 0;
    hipMalloc(&d_iterationCount, sizeof(unsigned long long));
    hipMemcpy(d_iterationCount, &h_iterationCount, sizeof(unsigned long long),
               hipMemcpyHostToDevice);

    SharedSize *d_AValues, *d_BValues;
    int aSize = sizeof(InPrec) * globalMmaShape.m * globalMmaShape.k;
    int bSize = sizeof(InPrec) * globalMmaShape.n * globalMmaShape.k;
    hipMalloc(&d_AValues, aSize);
    hipMalloc(&d_BValues, bSize);

    // Kind of a hack but we go to NaN if we let it keep incrementing
    int maxFloat = 32768;
    std::vector<half2> h_AValues{};
    // Fill the vector with increasing half-precision values
    // Note that this gets funny > 2048 because of imprecision of half values
    for (int m = 0; m < globalMmaShape.m; m++) {
        for (int k = 0; k < globalMmaShape.k; k += 2) {
            half2 val{};
            val.x = static_cast<half>(min(maxFloat, m * globalMmaShape.k + k));
            val.y = static_cast<half>(min(maxFloat, m * globalMmaShape.k + k + 1));
            h_AValues.push_back(val);
        }
    }

    PrintMatrix("Global A", reinterpret_cast<half*>(h_AValues.data()), globalMmaShape.m,
                globalMmaShape.k);

    std::vector<half2> h_BValues{};
    // Create identity matrix
    for (int row = 0; row < globalMmaShape.n; row++) {
        for (int col = 0; col < globalMmaShape.k; col += 2) {
            half2 val{0, 0};
            if (col == row)
                val.x = 1;
            else if (col + 1 == row)
                val.y = 1;
            h_BValues.push_back(val);
        }
    }
    PrintMatrix("Global B", reinterpret_cast<half*>(h_BValues.data()), globalMmaShape.n,
                globalMmaShape.k);

    hipMemcpy(d_AValues, h_AValues.data(), aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_BValues, h_BValues.data(), bSize, hipMemcpyHostToDevice);

    printf("Running kernel\n");

    // Each MMA operation is a 16x8x16 operation
    // There are 16x8 values calculated per warp of 32 threads
    // Each value requires 2 * 16 FLOPS due to the multiply and add
    // Total flops per warp per iteration is (16*8)*(2*16)=4096
    // Theoretical max for A100 is 312 TFLOPS
    // We have 4 Tensor cores per SM, 108 SM's, so 432 total TC's/GPU
    // This means each tensor core can do 312 TFLOPS / 432 TC's = 722 GFLOPS
    // If each operation is 4096 FLOP, then we would expect 176M mma operations per second
    hipEventRecord(start, 0);

    dim3 gridDim(ceil(1.0 * globalMmaShape.n / BlockMma::GetBlockTileDims().n),
                 ceil(1.0 * globalMmaShape.m / BlockMma::GetBlockTileDims().m), 1);
    dim3 blockDim(BlockMma::numWarps * WARPSIZE, 1, 1);
    size_t sharedMemBytes = BlockMma::pipelineDepth * BlockMma::ElemsPerStage * sizeof(SharedSize);
    printf("Requesting %lu bytes of shared memory\n", sharedMemBytes);
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(MmaPtxShared), hipFuncAttributeMaxDynamicSharedMemorySize,
                                   sharedMemBytes));
    MmaPtxShared<<<gridDim, blockDim, sharedMemBytes>>>(d_iterationCount, d_AValues, d_BValues,
                                                        globalMmaShape.k);

    gpuErrchk(hipEventRecord(stop, 0));

    gpuErrchk(hipEventSynchronize(stop));

    hipMemcpy(&h_iterationCount, d_iterationCount, sizeof(unsigned long long),
               hipMemcpyDeviceToHost);
    printf("Number of total iterations: %lld\n", h_iterationCount);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime /= 1000;

    printf("Kernel Elapsed time: %f seconds\n", elapsedTime);
    // Estimated TFLOPS that we computed
    const float tflops = static_cast<float>(globalMmaShape.m) * globalMmaShape.n *
                         globalMmaShape.k * 2 / elapsedTime / 1e12;
    printf("Estimated TFLOPS %.3f\n", tflops);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_iterationCount);
    hipFree(d_AValues);
    hipFree(d_BValues);
}

__global__ void MmaPtxShared(unsigned long long* iterationCount, SharedSize* AValues,
                             SharedSize* BValues, int kStride) {
    BlockMma::BlockTileMma(iterationCount, AValues, BValues, kStride);
}
