#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <hip/hip_vector_types.h>

#include <iostream>

#define WARPSIZE 32

// Low level namespace for fragments, and single mma operations
namespace Mma {
constexpr bool Debug = false;

struct mmaTileDims {
    int m{};
    int n{};
    int k{};
};

struct Coordinate {
    int row{};
    int col{};
};

// Declare a constant for reference elsewhere
// Dimensions of a fundamental mma operation in ptx
constexpr mmaTileDims dims{16, 8, 16};

// Represents one operand A, B, C, or D, of an mma operaetion that ldmatrix loads into registers.
// Each thread holds a piece of this operand. Input operands in half precision are packed in pairs
// into registers. Precision and fragment size dictate how many registers are required
template <typename T, int NumReg>
struct Fragment {
   public:
    T Registers[NumReg]{};

    __device__ void clear() {
        for (int j = 0; j < NumReg; j++) {
            Registers[j] = 0.0f;
        }
    }
};

// Making some aliases to make it easier to define specific functions taking only these template
// arguments due to the fact that they require inline ptx and aren't generic
using FragmentA_16x16 = Fragment<uint32_t, 4>;
using FragmentB_16x8 = Fragment<uint32_t, 2>;
using FragmentD_16x8 = Fragment<float, 4>;

__device__ inline uint32_t cvta_to_shared_u32(const void* pointer) {
    uint32_t address;
    // This is converting our generic 64 bit address to the shared memory state space. This
    // means subtracting the base address of the shared space, and then truncating to 32 bits
    // since shared memory all fits into 32 bits this is safe. The generic address space is 64
    // bits though.
    asm("{\n\t"
        "  .reg .u64 u64addr;\n\t"
        "  cvta.to.shared.u64 u64addr, %1;\n\t"
        "  cvt.u32.u64 %0, u64addr;\n\t"
        "}"
        : "=r"(address)
        : "l"(pointer));
    return address;
}

// Load row of shared memory into Fragment
// Since this has inline PTX it only works for a specific template specification
__device__ void loadAMatrix_16_16(const void* smem_row_start, FragmentA_16x16& A) {
    //  Page into A
    //  Pointer to 128 bit row of data in shared memory
    uint32_t smem_ptr;

    smem_ptr = cvta_to_shared_u32(smem_row_start);

    if (Debug) {
        if (threadIdx.x == 0) {
            printf("Shared 32b Memory Address A 0x%x\n", smem_ptr);
        }
    }

    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 "
        "{ %0, %1, %2, %3 }, [%4];"
        : "=r"(A.Registers[0]), "=r"(A.Registers[1]), "=r"(A.Registers[2]), "=r"(A.Registers[3])
        : "r"(smem_ptr));

    if (Debug) {
        // Inspect A
        for (int i = 0; i < 4; i++) {
            half2* tempVal = reinterpret_cast<half2*>(A.Registers[i]);
            printf("Thread %d, %d: A%d=%f, A%d=%f\n", threadIdx.x, threadIdx.y, i,
                   __half2float(tempVal->x), i, __half2float(tempVal->y));
        }
    }
}

// Load row of shared memory into Fragment
__device__ void loadBMatrix_16_8(const void* smem_row_start, FragmentB_16x8& B) {
    //  Page into A
    //  Pointer to 128 bit row of data in shared memory
    uint32_t smem_ptr;

    smem_ptr = cvta_to_shared_u32(smem_row_start);

    if (Debug) {
        if (threadIdx.x == 0) {
            printf("Shared 32b Memory Address B 0x%x\n", smem_ptr);
        }
    }

    // To get this to work out like the example, you don't want to transpose here.
    // It seems there is an implied transpose for the B matrix when you execute mma
    asm volatile(
        "ldmatrix.sync.aligned.x2.m8n8.shared.b16 "
        "{ %0, %1 }, [%2];"
        : "=r"(B.Registers[0]), "=r"(B.Registers[1])
        : "r"(smem_ptr));

    if (Debug) {
        // Inspect B
        for (int i = 0; i < 2; i++) {
            half2* tempVal = reinterpret_cast<half2*>(&B.Registers[i]);
            printf("Thread %d, %d: B%d=%f, B%d=%f\n", threadIdx.x, threadIdx.y, i,
                   __half2float(tempVal->x), i, __half2float(tempVal->y));
        }
    }
}

__device__ void mma_16_8_16(const FragmentA_16x16& A, const FragmentB_16x8& B,
                            const FragmentD_16x8& C, FragmentD_16x8& D) {
    // 16x8x8 TC Operation
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
        " { %0, %1, %2, %3 }, "
        " { %4, %5, %6, %7}, "
        " { %8, %9 }, "
        " { %10, %11, %12, %13 };"
        : "=f"(D.Registers[0]), "=f"(D.Registers[1]), "=f"(D.Registers[2]), "=f"(D.Registers[3])
        : "r"(A.Registers[0]), "r"(A.Registers[1]), "r"(A.Registers[2]), "r"(A.Registers[3]),
          "r"(B.Registers[0]), "r"(B.Registers[1]), "f"(C.Registers[0]), "f"(C.Registers[1]),
          "f"(C.Registers[2]), "f"(C.Registers[3]));

    if (Debug) {
        printf("Thread %d, %d: D0=%f, D1=%f, D2=%f, D3=%f\n", threadIdx.x, threadIdx.y,
               D.Registers[0], D.Registers[1], D.Registers[2], D.Registers[3]);
    }
}

// TODO, this feels like I'm hard coding this...maybe it can be templated?
// Compute the coordinate of a specific register
__device__ Coordinate GetDElementCoordinate_16_8_float(Coordinate& baseCoord, int threadInWarp,
                                                       int dIndex) {
    int row, col;
    // First 8x8 matrix
    if (dIndex < 2) {
        row = baseCoord.row + (threadInWarp / 4);
        col = baseCoord.col + ((threadInWarp % 4) * 2) + dIndex;
    }
    // second 8x8 matrix
    else {
        row = baseCoord.row + 8 + (threadInWarp / 4);
        col = baseCoord.col + ((threadInWarp % 4) * 2) + (dIndex / 2);
    }
    return {row, col};
}

};  // namespace Mma

// A single warp operation made up of many fragments of A, B, and D
namespace WarpMma {

// Warp  Parameters
constexpr int numAFragments = 4;
constexpr int numBFragments = 8;
constexpr int numDFragments = numAFragments * numBFragments;

struct WarpTileDims {
    int m{};
    int n{};
    int k{};
};

// TODO Rounding might not work out here, have to be careful
constexpr int numRegistersA =
    Mma::dims.m * Mma::dims.k * sizeof(half) / sizeof(uint32_t) / WARPSIZE;
constexpr int numRegistersB =
    Mma::dims.k * Mma::dims.n * sizeof(half) / sizeof(uint32_t) / WARPSIZE;
constexpr int numRegistersD =
    Mma::dims.m * Mma::dims.n * sizeof(float) / sizeof(uint32_t) / WARPSIZE;

__host__ __device__ constexpr WarpTileDims GetWarpTileDims() {
    int m = numAFragments * Mma::dims.m;
    int n = numBFragments * Mma::dims.n;
    // Warp handles a single k slice at a time in registers
    int k = Mma::dims.k;
    return WarpTileDims{m, n, k};
}

// Each WarpTile stores multiple operands A and B to compute a large output matrix D
// Template can specify the size of the actual warp tile.
struct WarpTile {
   public:
    Mma::Fragment<uint32_t, numRegistersA> A[numAFragments]{};
    Mma::Fragment<uint32_t, numRegistersB> B[numBFragments]{};
    Mma::Fragment<float, numRegistersD> D[numDFragments]{};

    // Given an WarpTile, clears the D registers to 0.0. Useful for when starting a computation
    __device__ void clearD() {
        for (int i = 0; i < numDFragments; i++) {
            D[i].clear();
        }
    }

    // TODO pass in k index here
    //  Given a WarpTile, loads all the A fragments into it
    __device__ void warpTileLoadA(half2* aTileAddr, Mma::Coordinate& baseCoord) {
        // Page fragment into A. This is 2 fragments
        for (int i = 0; i < numAFragments; i++) {
            // Upper left row
            int rowIndex = baseCoord.row + (i * Mma::dims.m);
            // TODO compute address for this thread
            Mma::loadAMatrix_16_16(aTileAddr, A[i]);
        }
    }

    // Given a WarpTile, loads all the B fragments into it
    __device__ void warpTileLoadB(half2* bTileAddr, Mma::Coordinate& baseCoord) {
        // Page fragment into B. This is 4 fragments
        // Need to duplicate addresses here for threads 16-31
        for (int i = 0; i < numBFragments; i++) {
            int colIndex = baseCoord.col + (i * Mma::dims.n);
            // TODO compute address for this thread
            Mma::loadBMatrix_16_8(bTileAddr, B[i]);
        }
    }

    __device__ int GetDIndex(const int aFragIndex, const int bFragIndex) {
        return aFragIndex * numBFragments + bFragIndex;
    }

    // Given a WarpTile with operands A and B, computes D=A*B+D
    __device__ void warpTileMma() {
        for (int a = 0; a < numAFragments; a++) {
            for (int b = 0; b < numBFragments; b++) {
                Mma::FragmentD_16x8& Dfrag = D[GetDIndex(a, b)];
                Mma::mma_16_8_16(A[a], B[b], Dfrag, Dfrag);
            }
        }
    }

    __device__ Mma::Coordinate GetBaseFragmentCoordinate(Mma::Coordinate& baseCoord,
                                                         const int aFragIndex,
                                                         const int bFragIndex) {
        int fragRow = baseCoord.row + (aFragIndex * Mma::dims.m);
        int fragCol = baseCoord.col + (bFragIndex * Mma::dims.n);
        return {fragRow, fragCol};
    }

    // Once computation is done, iterate over all output elements and apply epilogue
    __device__ void inspectResults(Mma::Coordinate& baseCoord) {
        for (int a = 0; a < numAFragments; a++) {
            for (int b = 0; b < numBFragments; b++) {
                Mma::Coordinate fragCoords = GetBaseFragmentCoordinate(baseCoord, a, b);
                Mma::FragmentD_16x8& Dfrag = D[GetDIndex(a, b)];
                for (int d = 0; d < numRegistersD; d++) {
                    int threadInWarp = threadIdx.x % WARPSIZE;
                    Mma::Coordinate elemCoord =
                        Mma::GetDElementCoordinate_16_8_float(fragCoords, threadInWarp, d);
                    // TODO perform addition of squared terms and comparison with epsilon here
                }
            }
        }
    }
};
};  // namespace WarpMma

namespace BlockMma {

// Block Parameters
constexpr int numWarpCols = 2;
constexpr int numWarpRows = 2;
constexpr int kSlices = 4;
constexpr int coarseFactor = 1;

struct BlockTileDims {
    int m{};
    int n{};
    int k{};
};

__host__ __device__ constexpr BlockTileDims GetBlockTileDims() {
    WarpMma::WarpTileDims warpDims = WarpMma::GetWarpTileDims();
    int m = numWarpRows * warpDims.m;
    int n = numWarpCols * warpDims.n;
    // Can buffer multiple k slices into shared memory at a time
    int k = kSlices * warpDims.k;
    return BlockTileDims{m, n, k};
}

// Compute how much shared memory to allocate when we launch the kernel
// Divide by two since it's an array of half2 values
constexpr BlockMma::BlockTileDims blockTileDims = GetBlockTileDims();
constexpr int aBlockTileSize = blockTileDims.m * blockTileDims.k / 2;
constexpr int bBlockTileSize = blockTileDims.n * blockTileDims.k / 2;

__device__ Mma::Coordinate GetBaseBlockCoordinate() {
    int baseRow = blockIdx.y * GetBlockTileDims().m;
    int baseCol = blockIdx.x * GetBlockTileDims().n;
    return {baseRow, baseCol};
}

__device__ Mma::Coordinate GetBaseWarpCoordinate(Mma::Coordinate baseBlockCoord, int warpId) {
    int warpRow = warpId / numWarpCols;
    int warpCol = warpId % numWarpCols;
    int baseRow = baseBlockCoord.row + warpRow * WarpMma::GetWarpTileDims().m;
    int baseCol = baseBlockCoord.col + warpCol * WarpMma::GetWarpTileDims().n;
    return {baseRow, baseCol};
}

__device__ void Mma(unsigned long long* iterationCount) {
    int tidx = threadIdx.x % 32;
    int warpId = threadIdx.x / 32;
    // Kind of a useless count to get compiler to not optimize away my code
    unsigned int count = 0;

    // We need 16 byte alignment here since LDMatrix will read rows of 16B at a time
    // Are static shared memory allocations already aligned?
    __shared__ __align__(16) half2 ATile[aBlockTileSize];
    __shared__ __align__(16) half2 BTile[bBlockTileSize];

    // Page Global --> Shared Memory

    // Compute Upper left coordinate that this block is responsible for
    Mma::Coordinate baseBlockCoord = GetBaseBlockCoordinate();
    // Compute the Upper left coordinate that each warp is responsible for
    Mma::Coordinate baseWarpCoord = GetBaseWarpCoordinate(baseBlockCoord, warpId);

    // At this point we are in the kernel, so cuda parallelism is taking place. We really have
    // numWarps running
    WarpMma::WarpTile warpTile;
    warpTile.clearD();

    // Accumulate into D as many times as we need to
    for (int kslice = 0; kslice < BlockMma::kSlices; kslice++) {
        warpTile.warpTileLoadA(ATile, baseWarpCoord);
        warpTile.warpTileLoadB(BTile, baseWarpCoord);
        warpTile.warpTileMma();
    }

    // TODO the warpTile should determime if values are in bounds
    // This is all here so everything isn't optimized away
    for (int i = 0; i < WarpMma::numDFragments; i++) {
        if (tidx == 0 && (warpTile.D[i].Registers[0] > 10.0f)) {
            count++;
        }
    }
    if (tidx == 0) {
        atomicAdd(iterationCount, count);
    }
}

};  // namespace BlockMma

__global__ void MmaPtxShared(unsigned long long* iterationCount);
__device__ uint get_smid(void);

constexpr bool Debug = false;

// ---------- Matrix Parameters ----------
constexpr int m = 1024;
constexpr int n = 1024;
constexpr int k = 1024;

// ---------- Mma parameters ----------
constexpr int totalFlopsPerOp = Mma::dims.m * Mma::dims.n * Mma::dims.k * 2;

// ---------- Warp parameters ----------

// ---------- Block parameters ----------
// How many warps to launch per block
constexpr int numWarps = BlockMma::numWarpCols * BlockMma::numWarpRows;

// ---------- Hardware parameters ----------

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Return the ID of the steaming multiprocesser this block is running on
__device__ uint get_smid(void) {
    uint ret;

    asm("mov.u32 %0, %smid;" : "=r"(ret));

    return ret;
}

int main(int argc, char* argv[]) {
    hipSetDevice(0);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate this just so kernel actually does something
    unsigned long long* d_iterationCount;
    unsigned long long h_iterationCount = 0;
    hipMalloc(&d_iterationCount, sizeof(unsigned long long));
    hipMemcpy(d_iterationCount, &h_iterationCount, sizeof(unsigned long long),
               hipMemcpyHostToDevice);

    printf("Running kernel\n");

    // Each MMA operation is a 16x8x16 operation
    // There are 16x8 values calculated per warp of 32 threads
    // Each value requires 2 * 16 FLOPS due to the multiply and add
    // Total flops per warp per iteration is (16*8)*(2*16)=4096
    // Theoretical max for A100 is 312 TFLOPS
    // We have 4 Tensor cores per SM, 108 SM's, so 432 total TC's/GPU
    // This means each tensor core can do 312 TFLOPS / 432 TC's = 722 GFLOPS
    // If each operation is 4096 FLOP, then we would expect 176M mma operations per second
    hipEventRecord(start, 0);

    dim3 gridDim(ceil(1.0 * n / BlockMma::GetBlockTileDims().n),
                 ceil(1.0 * m / BlockMma::GetBlockTileDims().m), 1);
    // 16 warps is the minimum to achieve 100% tensor core usage.
    // Interestingly, performance drops when you do 17 warps, likely because there are 4 tensor
    // cores, so we want a multiple of 4 warps for optimal performance.
    dim3 blockDim(WARPSIZE * numWarps, 1, 1);
    MmaPtxShared<<<gridDim, blockDim>>>(d_iterationCount);

    gpuErrchk(hipEventRecord(stop, 0));

    gpuErrchk(hipEventSynchronize(stop));

    hipMemcpy(&h_iterationCount, d_iterationCount, sizeof(unsigned long long),
               hipMemcpyDeviceToHost);
    printf("Number of total iterations: %lld\n", h_iterationCount);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime /= 1000;

    printf("Kernel Elapsed time: %f seconds\n", elapsedTime);
    // Estimated TFLOPS that we computed
    const float tflops = gridDim.x * blockDim.x / 32.0 * BlockMma::coarseFactor *
                         BlockMma::kSlices * WarpMma::numDFragments * totalFlopsPerOp /
                         elapsedTime / 1e12;
    printf("Estimated TFLOPS %.3f\n", tflops);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_iterationCount);
}

__global__ void MmaPtxShared(unsigned long long* iterationCount) { BlockMma::Mma(iterationCount); }
